#include "hip/hip_runtime.h"
#include "matrix.hpp"
#include <cstddef>
#include <iostream>

__global__
void init(MatrixValType* matrix, const MatrixSize size, const MatrixValType val) {
  std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= size.height * size.width) return;
  
  matrix[i] = val;
}

Matrix::Matrix(const MatrixSize& size) {
  CUDA_CALL(hipMalloc(&this->gpuData, size.height * size.width * sizeof(MatrixValType)));
  this->size = size;
}

Matrix::Matrix(const MatrixSize& size, const MatrixValType val) : Matrix(size) {
  init<<<(size.height * size.width) / 32 + 1, 32>>>(this->gpuData, this->size, val);
  CUDA_CALL(hipGetLastError());
  CUDA_CALL(hipDeviceSynchronize());
}

void Matrix::show() const {
  MatrixValType* val = new MatrixValType[size.width * size.height];
  hipMemcpy(val, this->gpuData, size.height * size.width * sizeof(MatrixValType), hipMemcpyDeviceToHost);

  for (int i = 0; i < size.height; i++) {
    for (int j = 0; j < size.height; j++) {
      std::cout << " " << val[i*size.height + j] << " ";
    }
    std::cout << "\n";
  }
}
