#include "hip/hip_runtime.h"
#include "matrix.hpp"
#include <cstddef>
#include <iostream>

__global__ void init_gpu(MatrixValType *matrix, MatrixSize size,
                         const MatrixValType val) {
  const auto i = (blockIdx.x * blockDim.x + threadIdx.x);

  if (i >= size.total)
    return;

  matrix[i] = val;
}

__global__ void multiply_gpu(MatrixValType *matrix, const MatrixSize size,
                             const MatrixValType scalar) {
  std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= size.total)
    return;

  matrix[i] = matrix[i] * scalar;
}

__global__ void multiply_gpu(MatrixValType *in1, const MatrixSize in1Size,
                             MatrixValType *in2, const MatrixSize in2Size,
                             MatrixValType *out, const MatrixSize outSize) {
  std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  std::size_t y = i / outSize.width;
  std::size_t x = i - (y * outSize.width);

  if (i >= outSize.total)
    return;

  out[y * outSize.height + x] = 0;

  for (int j = 0; j < in1Size.width; j++) {
    out[y * outSize.width + x] +=
        in1[y * in1Size.width + j] * in2[j * in2Size.width + x];
  }
}

Matrix::~Matrix() { CUDA_CALL(hipFree(this->gpuData)); }

Matrix::Matrix(const MatrixSize &size) : size(size) {
  CUDA_CALL(hipMalloc(&this->gpuData, size.total * sizeof(MatrixValType)));
}

Matrix::Matrix(const MatrixSize &size, const MatrixValType val) : Matrix(size) {
  init_gpu<<<size.total / 32 + 1, 32>>>(this->gpuData, size, val);
  CUDA_CALL(hipGetLastError());
  CUDA_CALL(hipDeviceSynchronize());
}

Matrix::Matrix(const Matrix &copied) : size(copied.size) {
  MatrixValType *copied_gpu_data;
  hipMalloc(&copied_gpu_data, copied.size.total * sizeof(MatrixValType));
  hipMemcpy(copied_gpu_data, copied.gpuData,
             size.total * sizeof(MatrixValType), hipMemcpyDeviceToDevice);
  this->gpuData = copied_gpu_data;
}

void Matrix::multiply(const MatrixValType scalar) {
  multiply_gpu<<<size.total / 32 + 1, 32>>>(this->gpuData, this->size, scalar);
  CUDA_CALL(hipGetLastError());
  CUDA_CALL(hipDeviceSynchronize());
}

Matrix Matrix::multiply(const Matrix &other) const {
  Matrix out(MatrixSize(size.height, other.size.width));

  multiply_gpu<<<size.total / 32 + 1, 32>>>(gpuData, size, other.gpuData,
                                            other.size, out.gpuData, out.size);
  CUDA_CALL(hipGetLastError());
  CUDA_CALL(hipDeviceSynchronize());

  return out;
}

void Matrix::show() const {
  MatrixValType *val = new MatrixValType[size.total];
  hipMemcpy(val, this->gpuData, size.total * sizeof(MatrixValType),
             hipMemcpyDeviceToHost);

  for (int i = 0; i < size.height; i++) {
    for (int j = 0; j < size.width; j++) {
      std::cout << " " << val[i * size.width + j] << " ";
    }
    std::cout << "\n";
  }
}
