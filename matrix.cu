#include "hip/hip_runtime.h"
#include "cuda.hpp"
#include "matrix.hpp"

/* Contructor and destructor */

__global__ void init_gpu(MatrixValType *matrix, MatrixSize size,
                         const MatrixValType val) {
  const auto i = (blockIdx.x * blockDim.x + threadIdx.x);

  if (i >= size.total)
    return;

  matrix[i] = val;
}

Matrix::~Matrix() { CUDA_CALL(hipFree(this->gpuData)); }

Matrix::Matrix(const MatrixSize &size) : size(size) {
  CUDA_CALL(
      hipMalloc(&this->gpuData, this->size.total * sizeof(MatrixValType)));
}

Matrix::Matrix(const MatrixSize &size, const MatrixValType val) : Matrix(size) {
  init_gpu<<<this->groupSize(), this->threadSize()>>>(this->gpuData, this->size,
                                                      val);
  CUDA_CALL(hipGetLastError());
  CUDA_CALL(hipDeviceSynchronize());
}

Matrix::Matrix(const Matrix &copied) : size(copied.size) {
  MatrixValType *copied_gpu_data;

  hipMalloc(&copied_gpu_data, copied.size.total * sizeof(MatrixValType));
  hipMemcpy(copied_gpu_data, copied.gpuData,
             this->size.total * sizeof(MatrixValType),
             hipMemcpyDeviceToDevice);

  this->gpuData = copied_gpu_data;
}

/* Misc */

CPUMatrix Matrix::toCPU() const {
  CPUMatrix matrix;
  matrix.reserve(this->size.height);

  for (int y = 0; y < size.height; y++) {
    matrix.push_back(std::vector<MatrixValType>());
    matrix[y].resize(size.width);

    hipMemcpy(&(*matrix[y].begin()), &gpuData[y * size.width],
               size.width * sizeof(MatrixValType), hipMemcpyDeviceToHost);
  }

  return matrix;
}

Matrix Matrix::fromCPU(const CPUMatrix &input) {
  Matrix m(MatrixSize(input.size(), input[0].size()));

  hipMalloc(&m.gpuData, m.size.total * sizeof(MatrixValType));

  for (std::size_t y = 0; y < input.size(); y++) {
    hipMemcpy(&m.gpuData[y * m.size.width], &(*input[y].begin()),
               m.size.width * sizeof(MatrixValType), hipMemcpyHostToDevice);
  }

  return m;
}

// TODO: This should be moved to CPUMatrix function
void Matrix::show() const {
  const auto matrix = this->toCPU();

  for (std::size_t i = 0; i < matrix.size(); i++) {
    for (std::size_t j = 0; j < matrix[0].size(); j++) {
      std::cout << " " << matrix[i][j] << " ";
    }
    std::cout << "\n";
  }
}

/* Adding */

__global__ void add_gpu(MatrixValType *in, MatrixValType *other,
                        MatrixValType *out, const MatrixSize size) {
  std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= size.total)
    return;

  out[i] = in[i] + other[i];
}

void Matrix::add(const Matrix &other, Matrix &result) const {
  if (result.size.width != this->size.width ||
      other.size.width != this->size.width)
    throw new InvalidMatrixSize("Input Matrix width is not valid");

  if (result.size.height != this->size.height ||
      other.size.height != this->size.height)
    throw new InvalidMatrixSize("Input Matrix height is not valid");

  add_gpu<<<size.total / 32 + 1, 32>>>(this->gpuData, other.gpuData,
                                       result.gpuData, this->size);

  CUDA_CALL(hipGetLastError());
  CUDA_CALL(hipDeviceSynchronize());
}

Matrix Matrix::add(const Matrix &other) const {
  Matrix out(MatrixSize(this->size.height, this->size.width));
  this->add(other, out);
  return out;
}

__global__ void add_gpu(MatrixValType *in, MatrixValType scalar,
                        MatrixValType *out, const MatrixSize size) {
  const std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= size.total)
    return;

  out[i] = in[i] + scalar;
}

void Matrix::add(const MatrixValType scalar, Matrix &result) const {
  if (result.size.width != this->size.width)
    throw new InvalidMatrixSize("Result Matrix width is not valid");

  if (result.size.height != this->size.height)
    throw new InvalidMatrixSize("Result Matrix height is not valid");

  add_gpu<<<size.total / 32 + 1, 32>>>(this->gpuData, scalar, result.gpuData,
                                       this->size);

  CUDA_CALL(hipGetLastError());
  CUDA_CALL(hipDeviceSynchronize());
}

Matrix Matrix::add(const MatrixValType scalar) const {
  Matrix out(MatrixSize(this->size.height, this->size.width));
  this->add(scalar, out);
  return out;
}

/* Multiplication */

__global__ void multiply_gpu(MatrixValType *in, const MatrixValType scalar,
                             MatrixValType *out, const MatrixSize size) {
  std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= size.total)
    return;

  out[i] = in[i] * scalar;
}

void Matrix::multiply(const MatrixValType scalar, Matrix &out) const {
  if (this->size.height != out.size.height)
    throw new InvalidMatrixSize(
        "Current matrix height does not match result matrix height");

  if (this->size.width != out.size.width)
    throw new InvalidMatrixSize(
        "Current matrix width does not match result matrix width");

  multiply_gpu<<<size.total / 32 + 1, 32>>>(this->gpuData, scalar, out.gpuData,
                                            this->size);
  CUDA_CALL(hipGetLastError());
  CUDA_CALL(hipDeviceSynchronize());
}

Matrix Matrix::multiply(const MatrixValType scalar) const {
  Matrix out(this->size);
  this->multiply(scalar, out);
  return out;
}

__global__ void multiply_gpu(const MatrixValType *in1, const MatrixSize in1Size,
                             const MatrixValType *in2, const MatrixSize in2Size,
                             MatrixValType *out, const MatrixSize outSize) {
  std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  std::size_t y = i / outSize.width;
  std::size_t x = i - (y * outSize.width);

  if (i >= outSize.total)
    return;

  MatrixValType tmp = 0;
  
  for (int j = 0; j < in1Size.width; j++) {
    tmp += in1[y * in1Size.width + j] * in2[j * in2Size.width + x];
  }

  out[y * outSize.width + x] = tmp;
}

void Matrix::multiply(const Matrix &other, Matrix &out) const {
  if (this->size.width != other.size.height)
    throw new InvalidMatrixSize(
        "Current matrix width does not match other matrix height");

  if (this->size.height != out.size.height)
    throw new InvalidMatrixSize(
        "Current matrix height does not match result matrix height");

  if (other.size.width != out.size.width)
    throw new InvalidMatrixSize(
        "Other matrix width does not match result matrix width");

  multiply_gpu<<<size.total / 32 + 1, 32>>>(this->gpuHandle(), this->getSize(),
                                            other.gpuHandle(), other.getSize(),
                                            out.gpuData, out.size);
  CUDA_CALL(hipGetLastError());
  CUDA_CALL(hipDeviceSynchronize());
}

Matrix Matrix::multiply(const Matrix &other) const {
  Matrix out(MatrixSize(this->size.height, other.size.width));
  this->multiply(other, out);
  return out;
}

/* Transposition */

__global__ void transpose_gpu(const MatrixValType *in, const MatrixSize inSize,
                              MatrixValType *out, const MatrixSize outSize) {
  std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  std::size_t y = i / outSize.width;
  std::size_t x = i - (y * outSize.width);

  if (i >= outSize.total)
    return;

  out[y * outSize.width + x] = in[x * inSize.width + y];
}

void Matrix::transpose(Matrix& result) const {
  if (this->getSize().width != result.getSize().height
      ||
      this->getSize().height != result.getSize().width)
    throw new InvalidMatrixSize("Result Matrix does not have proper size");

  transpose_gpu<<<size.total / 32 + 1, 32>>>(
      this->gpuHandle(), this->getSize(), result.gpuHandle(), result.getSize());
  CUDA_CALL(hipGetLastError());
  CUDA_CALL(hipDeviceSynchronize());
}

Matrix Matrix::transpose() const {
  Matrix result(MatrixSize(this->size.width, this->size.height));
  this->transpose(result);
  return result;
}
