#include "hip/hip_runtime.h"


#include <iostream>

#define CUDA_CALL(x)							\
  {                                                                            \
    hipError_t cuda_error__ = (x);                                            \
    if (cuda_error__)                                                          \
      std::cout << "CUDA error: " #x " returned "                              \
                << hipGetErrorString(cuda_error__) << std::endl;              \
  }


__global__
void testKernel() {
  return;
}


void runKernel() {
  testKernel<<<1, 1>>>();
  CUDA_CALL(hipDeviceSynchronize());
}
